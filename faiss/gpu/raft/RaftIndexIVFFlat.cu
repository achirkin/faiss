#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/IndexFlat.h>
#include <faiss/IndexIVFFlat.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuIndexIVFFlat.h>
#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/raft/RaftIndexIVFFlat.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/CopyUtils.cuh>
#include <faiss/gpu/utils/Float16.cuh>

#include <raft/spatial/knn/ivf_flat.cuh>

#include <limits>

namespace faiss {
namespace gpu {

RaftIndexIVFFlat::RaftIndexIVFFlat(
        GpuResourcesProvider* provider,
        const faiss::IndexIVFFlat* index,
        GpuIndexIVFFlatConfig config)
        : GpuIndexIVFFlat(provider, index, config),
          raft_handle(resources_->getDefaultStream(config_.device)) {
    copyFrom(index);
}

RaftIndexIVFFlat::RaftIndexIVFFlat(
        GpuResourcesProvider* provider,
        int dims,
        int nlist,
        faiss::MetricType metric,
        GpuIndexIVFFlatConfig config)
        : GpuIndexIVFFlat(provider, dims, nlist, metric, config),
          raft_handle(resources_->getDefaultStream(config_.device)) {}

RaftIndexIVFFlat::~RaftIndexIVFFlat() {
    RaftIndexIVFFlat::reset();
}

void RaftIndexIVFFlat::copyFrom(const faiss::IndexIVFFlat* index) {
    DeviceScope scope(config_.device);
    GpuIndex::copyFrom(index);
    FAISS_ASSERT(index->nlist > 0);
    FAISS_THROW_IF_NOT_FMT(
            index->nlist <= (Index::idx_t)std::numeric_limits<int>::max(),
            "GPU index only supports %zu inverted lists",
            (size_t)std::numeric_limits<int>::max());
    FAISS_THROW_IF_NOT_FMT(
            index->nprobe > 0 && index->nprobe <= getMaxKSelection(),
            "GPU index only supports nprobe <= %zu; passed %zu",
            (size_t)getMaxKSelection(),
            index->nprobe);

//    FAISS_ASSERT(metric_type != faiss::METRIC_L2 &&
//                 metric_type != faiss::METRIC_INNER_PRODUCT);
//
//    if (!index->is_trained) {
//        // copied in GpuIndex::copyFrom
//        FAISS_ASSERT(!is_trained && ntotal == 0);
//        return;

//    }
//
//    // copied in GpuIndex::copyFrom
//    // ntotal can exceed max int, but the number of vectors per inverted
//    // list cannot exceed this. We check this in the subclasses.
//    FAISS_ASSERT(is_trained && (ntotal == index->ntotal));
//
//    // Since we're trained, the quantizer must have data
//    FAISS_ASSERT(index->quantizer->ntotal > 0);
//
//
    /**
     * TODO: Copy centers and center norms from quantizer
     * Things to do:
     *    1. Copy index_->quantizer->vectors_ to raft_index->centers
     *    2. Copy index_->quantizer->norms_ to raft_index->center_norms
     */
    /**
     * TODO: Copy IVF data, indices, list_sizes, list_offsets from index->invlists
     *
     * Things to do:
     *    1. index->ivflists->data() is going to need to be translated over to our format
     *       (even the interleaved format is a little different)
     *
     *       The GpuIndexIVFFlat has a function translateCodesToGpu_() for this
     *
     *    2. We will need to copy  list_sizes, indices, and list_offsets
     */
    if (index->is_trained) {
        // TODO: A proper copy of the index without retraining
        // For now, just get all the data from the index, and train our index
        // anew.
        FAISS_ASSERT(index->d == this->d);
        FAISS_ASSERT(index->metric_arg == this->metric_arg);
        FAISS_ASSERT(index->metric_type == this->metric_type);
        FAISS_ASSERT(index->nlist == this->nlist);

        Index::idx_t quantizer_ntotal = index->quantizer->ntotal;
        Index::idx_t index_ntotal = index->ntotal;

        std::cout << "Calling copyFrom with trained index with "  << quantizer_ntotal << " items" << std::endl;
        auto stream = raft_handle.get_stream();

        auto total_elems = size_t(quantizer_ntotal) * size_t(index->quantizer->d);
        rmm::device_uvector<float> buf_dev(total_elems, stream);
        {
            std::vector<float> buf_host(total_elems);
            index->quantizer->reconstruct_n(0, quantizer_ntotal, buf_host.data());
            raft::copy(buf_dev.data(), buf_host.data(), total_elems, stream);
        }

        // build an empty index (because we don't have a constructor that doesn't train).
        RaftIndexIVFFlat::train(quantizer_ntotal, buf_dev.data());
        // override centers
        raft::copy(
            this->raft_knn_index->centers().data_handle(),
            _,
            this->nlist * this->d,
            stream
        );
        // add data
        if(index_ntotal > 0) {
            std::cout << "Adding " << index_ntotal << " vectors to index" << std::endl;
            total_elems = size_t(index_ntotal) * size_t(index->d);
            buf_dev.resize(total_elems, stream);
            {
                std::vector<float> buf_host(total_elems);
                index->reconstruct_n(0, index->ntotal, buf_host.data());
                raft::copy(buf_dev.data(), buf_host.data(), total_elems, stream);
            }
            RaftIndexIVFFlat::addImpl_(index_ntotal, buf_dev.data(), nullptr);
        }
    } else {
        // index is not trained, so we can remove ours as well (if there was
        // any)
        std::cout << "Calling copyFrom with index that hasn't been trained" << std::endl;
        raft_knn_index.reset();
    }
    this->is_trained = index->is_trained;
}

void RaftIndexIVFFlat::reserveMemory(size_t numVecs) {
    std::cout << "Reserving memory for " << numVecs << " vectors." << std::endl;
    reserveMemoryVecs_ = numVecs;
    if (raft_knn_index.has_value()) {
        DeviceScope scope(config_.device);

        // TODO: Need to figure out if this is absolutely necessary.

        /**
         * For example:
         * raft::spatial::knn::ivf_flat::allocate_ivf_lists(
         *      raft_handle, *raft_knn_index, numVecs);
         *
         * raft::spatial::knn::ivf_flat::populate(
         *      raft_handle, *raft_knn_index,
         *      n_centroids, centroids,
         *      n_vectors, ivf);
         *
         */
    }
}

size_t RaftIndexIVFFlat::reclaimMemory() {
    std::cout << "Reclaiming memory" << std::endl;

    // TODO: Need to figure out if this is absolutely necessary
    /**
     * For example:
     * raft::spatial::knn::ivf_flat::reclaim_ivf_lists(
     *      raft_handle, *raft_knn_index, numVecs);
     */
    return 0;
}

void RaftIndexIVFFlat::train(Index::idx_t n, const float* x) {
    DeviceScope scope(config_.device);

    std::cout << "Calling train() with " << n << " rows" << std::endl;
    if (this->is_trained) {
        FAISS_ASSERT(raft_knn_index.has_value());
        return;
    }

    raft::spatial::knn::ivf_flat::index_params raft_idx_params;
    raft_idx_params.n_lists = nlist;
    raft_idx_params.add_data_on_build = false;
    raft_idx_params.kmeans_trainset_fraction = 1.0;
    switch (this->metric_type) {
        case faiss::METRIC_L2:
            raft_idx_params.metric = raft::distance::DistanceType::L2Expanded;
            break;
        case faiss::METRIC_INNER_PRODUCT:
            raft_idx_params.metric = raft::distance::DistanceType::InnerProduct;
            break;
        default:
            FAISS_THROW_MSG("Metric is not supported.");
    }

    raft_knn_index.emplace(
        raft::spatial::knn::ivf_flat::build(raft_handle, raft_idx_params,
                                            const_cast<float*>(x),
                                            n, (faiss::Index::idx_t)d));

    raft_handle.sync_stream();
    this->is_trained = true;
}

int RaftIndexIVFFlat::getListLength(int listId) const {
    FAISS_ASSERT(raft_knn_index.has_value());
    DeviceScope scope(config_.device);

    uint32_t size;
    raft::copy(&size, raft_knn_index->list_sizes().data_handle() + listId,
               1, raft_handle.get_stream());
    raft_handle.sync_stream();
    return int(size);
}

std::vector<uint8_t> RaftIndexIVFFlat::getListVectorData(
        int listId,
        bool gpuFormat) const {
    FAISS_ASSERT(raft_knn_index.has_value());
    DeviceScope scope(config_.device);

    std::cout << "Calling getListVectorData for " << listId << std::endl;

    using elem_t = decltype(raft_knn_index->data())::element_type;
    size_t dim = raft_knn_index->dim();
    Index::idx_t offsets[2];
    raft::copy(offsets, raft_knn_index->list_offsets().data_handle() + listId, 2, raft_handle.get_stream());

    raft_handle.sync_stream();
    size_t byte_offset = offsets[0] * sizeof(elem_t) * dim;
    // the interleaved block can be slightly larger than the list size (it's
    // rounded up)
    size_t byte_size = size_t(offsets[1]) * sizeof(elem_t) * dim - byte_offset;
    std::vector<uint8_t> vec(byte_size);
    raft::copy(
            vec.data(),
            reinterpret_cast<const uint8_t*>(raft_knn_index->data().data_handle()) +
                    byte_offset,
            byte_size,
            raft_handle.get_stream());
    return vec;
}

void RaftIndexIVFFlat::reset() {
    raft_knn_index.reset();
    this->ntotal = 0;
}

std::vector<Index::idx_t> RaftIndexIVFFlat::getListIndices(int listId) const {
    FAISS_ASSERT(raft_knn_index.has_value());
    DeviceScope scope(config_.device);

    Index::idx_t offset;
    uint32_t size;

    raft::copy(&offset, raft_knn_index->list_offsets().data_handle() + listId, 1, raft_handle.get_stream());
    raft::copy(&size, raft_knn_index->list_sizes().data_handle() + listId, 1, raft_handle.get_stream());
    raft_handle.sync_stream();

    std::vector<Index::idx_t> vec(size);
    raft::copy(
            vec.data(),
            raft_knn_index->indices().data_handle() + offset,
            size,
            raft_handle.get_stream());
    return vec;
}

void RaftIndexIVFFlat::addImpl_(
        int n,
        const float* x,
        const Index::idx_t* xids) {
    // Device is already set in GpuIndex::add
    FAISS_ASSERT(is_trained);
    FAISS_ASSERT(n > 0);

//    // Not all vectors may be able to be added (some may contain NaNs etc)
//    index_->addVectors(data, labels);

    // but keep the ntotal based on the total number of vectors that we
    // attempted to add

    std::cout << "Calling addImpl_ with " << n << " vectors." << std::endl;
    raft::spatial::knn::ivf_flat::extend(raft_handle, *raft_knn_index, x, xids, (Index::idx_t)n);
    this->ntotal += n;
}

void RaftIndexIVFFlat::searchImpl_(
        int n,
        const float* x,
        int k,
        float* distances,
        Index::idx_t* labels) const {
    // Device is already set in GpuIndex::search
    FAISS_ASSERT(raft_knn_index.has_value());
    FAISS_ASSERT(n > 0);
    FAISS_THROW_IF_NOT(nprobe > 0 && nprobe <= nlist);

    std::cout << "Calling searchImpl_ with " << n << " rows" << std::endl;
    raft::spatial::knn::ivf_flat::search_params pams;
    pams.n_probes = nprobe;
    raft::spatial::knn::ivf_flat::search<float, faiss::Index::idx_t>(
            raft_handle,
            pams,
            *raft_knn_index,
            const_cast<float*>(x),
            static_cast<std::uint32_t>(n),
            static_cast<std::uint32_t>(k),
            labels,
            distances);

    raft_handle.sync_stream();
}

void RaftIndexIVFFlat::rebuildRaftIndex(const float* x, Index::idx_t n_rows) {
    raft::spatial::knn::ivf_flat::index_params pams;

    std::cout << "Calling rebuildRaftIndex with " << n_rows << " rows" << std::endl;
    pams.n_lists = this->nlist;
    switch (this->metric_type) {
        case faiss::METRIC_L2:
            pams.metric = raft::distance::DistanceType::L2Expanded;
            break;
        case faiss::METRIC_INNER_PRODUCT:
            pams.metric = raft::distance::DistanceType::InnerProduct;
            break;
        default:
            FAISS_THROW_MSG("Metric is not supported.");
    }
    pams.metric_arg = this->metric_arg;
    pams.kmeans_trainset_fraction = 1.0;
    pams.add_data_on_build = false;

    raft_knn_index.emplace(raft::spatial::knn::ivf_flat::build(
            this->raft_handle, pams, x, n_rows, uint32_t(this->d)));

    this->raft_handle.sync_stream();
    this->is_trained = true;
    this->ntotal = n_rows;
}

} // namespace gpu
} // namespace faiss
